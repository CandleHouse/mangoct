#include "hip/hip_runtime.h"
#include "FpjClass_Agent.cuh"

#define PI 3.1415926536f

__global__ void InitU(float* u, const int N, const float du, const float offcenter)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < N)
	{
		u[tid] = (tid - (N - 1) / 2.0f) * du + offcenter;
	}
}

__global__ void InitBeta(float* beta, const int V)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid<V)
	{
		beta[tid] = (360.0f / V * tid) * PI / 180;
	}
}

void InitializeU_Agent(float* &u, const int N, const float du, const float offcenter)
{
	if (u != nullptr)
		hipFree(u);

	hipMalloc((void**)&u, N * sizeof(float));
	InitU << <(N + 511) / 512, 512 >> > (u, N, du, offcenter);
}

void InitializeBeta_Agent(float *& beta, const int V)
{
	if (beta != nullptr)
		hipFree(beta);

	hipMalloc((void**)&beta, V * sizeof(float));
	InitBeta << < (V + 511) / 512, 512 >> > (beta, V);
}

void MallocManaged_Agent(float * &p, const int size)
{
	hipMallocManaged((void**)&p, size);
}

void FreeMemory_Agent(float* &p)
{
	hipFree(p);
	p = nullptr;
}
